
#include <hip/hip_runtime.h>
/**
 * @file device_storage.cu
 * @brief Device storage implementation for global constant arrays (deprecated)
 *
 * @details
 * This file previously defined __device__ constant memory arrays that were
 * shared across compilation units. These arrays have been migrated to regular
 * GPU memory (cudaMalloc) to support non-RDC compilation mode.
 *
 * @history Migration timeline:
 *
 * 1. CRAM T Matrix (2025-10-16)
 *    - Previous: __device__ float T_const[N_NUCLIDES * N_NUCLIDES];
 *    - Issue: "invalid device symbol" error in non-RDC mode
 *    - Solution: Migrated to LDM::d_T_matrix via cudaMalloc()
 *    - Impact: All decay calculations now use ks.T_matrix pointer
 *    - Files affected: 4 kernel files, ldm_cram2.cu, ldm_func_simulation.cu
 *
 * 2. Flex Height Levels (2025-10-16)
 *    - Previous: __device__ float d_flex_hgt[50];
 *    - Issue: "invalid device symbol" + "illegal memory access" in EKI mode
 *    - Solution: Migrated to LDM::d_flex_hgt via cudaMalloc()
 *    - Impact: All vertical interpolation now uses ks.flex_hgt pointer
 *    - Files affected: 4 kernel files, ldm_mdata_loading.cu, ldm_mdata_cache.cu
 *
 * @architecture Old vs. New:
 *
 * Old (RDC mode):
 * @code
 *   // device_storage.cu
 *   __device__ float d_flex_hgt[50];
 *
 *   // Some other file
 *   extern __device__ float d_flex_hgt[];
 *   cudaMemcpyToSymbol(d_flex_hgt, h_data, size);
 *
 *   // Kernel
 *   __global__ void kernel() {
 *       float height = d_flex_hgt[idx];  // Direct access
 *   }
 * @endcode
 *
 * New (non-RDC mode):
 * @code
 *   // ldm.cuh
 *   class LDM {
 *       float* d_flex_hgt;
 *   };
 *
 *   // ldm.cu
 *   LDM::LDM() {
 *       cudaMalloc(&d_flex_hgt, 50 * sizeof(float));
 *   }
 *
 *   // Kernel call site
 *   ks.flex_hgt = d_flex_hgt;
 *   kernel<<<blocks, threads>>>(ks);
 *
 *   // Kernel
 *   __global__ void kernel(KernelScalars ks) {
 *       float height = ks.flex_hgt[idx];  // Pointer access
 *   }
 * @endcode
 *
 * @benefits Non-RDC mode:
 * - Smaller binaries (~30% reduction)
 * - Faster compilation (no device link stage)
 * - Better CUDA toolkit compatibility
 * - Simpler build system
 * - Easier debugging
 * - No "invalid device symbol" errors
 *
 * @note This file is retained for documentation purposes
 * @note Can be safely deleted once all references removed
 * @note Does not contribute to compilation (no actual code)
 *
 * @author Juryong Park
 * @date 2025-10-16 (Non-RDC migration)
 * @see src/core/ldm.cuh for new memory management
 * @see src/core/params.hpp for KernelScalars definition
 */

// ============================================================================
// REMOVED: All __device__ constant memory arrays
// ============================================================================
//
// This file previously contained __device__ declarations for:
// - d_flex_hgt[50]: Vertical height levels for meteorological interpolation
// - T_const[N*N]: CRAM decay transition matrix for radioactive decay chains
//
// All arrays have been migrated to regular GPU memory (cudaMalloc) and are
// now managed by the LDM class. See header file for migration details.
//
// ============================================================================
