// ldm_eki_reader.cu - Implementation of EKI IPC Reader
#include "ldm_eki_reader.cuh"
#include "../debug/memory_doctor.cuh"
#include <numeric>
#include <algorithm>

namespace LDM_EKI_IPC {

// ============================================================================
// Constructor / Destructor
// ============================================================================

EKIReader::EKIReader()
    : config_fd(-1), data_fd(-1), config_map(nullptr),
      data_map(nullptr), data_size(0), initialized(false) {
}

EKIReader::~EKIReader() {
    cleanup();
}

// ============================================================================
// Wait for Ensemble Data
// ============================================================================

bool EKIReader::waitForEnsembleData(int timeout_seconds, int expected_iteration) {
    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Waiting for ensemble data from Python (timeout: "
              << Color::BOLD << timeout_seconds << "s" << Color::RESET << ")...\n";

    const char* config_path = "/dev/shm/ldm_eki_ensemble_config";
    const char* data_path = "/dev/shm/ldm_eki_ensemble_data";

    // Store last iteration ID to detect fresh data
    static int last_iteration_id = -1;

    // Wait for config file to appear with fresh data
    for (int i = 0; i < timeout_seconds; i++) {
        if (access(config_path, F_OK) == 0 && access(data_path, F_OK) == 0) {
            // Read config to check iteration ID
            int config_fd = open(config_path, O_RDONLY);
            if (config_fd >= 0) {
                EnsembleConfig config;
                ssize_t bytes_read = read(config_fd, &config, sizeof(config));
                close(config_fd);

                if (bytes_read == sizeof(config)) {
                    // Check if this is new data (different iteration ID)
                    if (config.timestep_id > last_iteration_id) {
                        // New iteration detected, now check if data is ready
                        int test_fd = open(data_path, O_RDONLY);
                        if (test_fd >= 0) {
                            EnsembleDataHeader header;
                            bytes_read = read(test_fd, &header, sizeof(header));
                            close(test_fd);

                            if (bytes_read == sizeof(header) && header.status == 1) {
                                std::cout << Color::GREEN << "✓ " << Color::RESET
                                          << "Fresh ensemble data detected (iteration " << Color::BOLD
                                          << config.timestep_id << Color::RESET << ")\n";
                                last_iteration_id = config.timestep_id;
                                return true;
                            }
                        }
                    } else if (config.timestep_id == last_iteration_id && i > 5) {
                        // Same iteration ID after 5 seconds - probably stale data
                        if (i % 5 == 0) {
                            std::cout << "\rWaiting for new data... (iteration " << config.timestep_id << ")" << std::flush;
                        }
                    }
                }
            }
        }
        sleep(1);
    }

    std::cerr << Color::RED << Color::BOLD << "[ERROR] " << Color::RESET
              << "Timeout waiting for ensemble data\n";
    return false;
}

// ============================================================================
// Read Configuration
// ============================================================================

bool EKIReader::readEnsembleConfig(int& num_states, int& num_ensemble, int& timestep_id) {
    const char* shm_path = "/dev/shm/ldm_eki_ensemble_config";

    config_fd = open(shm_path, O_RDONLY);
    if (config_fd < 0) {
        fprintf(stderr, "%s[ERROR]%s ", Color::RED, Color::RESET); perror(" Failed to open config");
        return false;
    }

    EnsembleConfig config;
    ssize_t bytes_read = read(config_fd, &config, sizeof(config));
    close(config_fd);
    config_fd = -1;

    if (bytes_read != sizeof(config)) {
        std::cerr << Color::RED << "[ERROR] " << Color::RESET << "Failed to read config (got " << bytes_read << " bytes)" << std::endl;
        return false;
    }

    num_states = config.num_states;
    num_ensemble = config.num_ensemble;
    timestep_id = config.timestep_id;

    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Config loaded: " << Color::BOLD << num_states << "×" << num_ensemble << Color::RESET
              << " (timestep " << timestep_id << ")\n";
    return true;
}

// ============================================================================
// Read Ensemble States
// ============================================================================

bool EKIReader::readEnsembleStates(std::vector<float>& output, int& num_states, int& num_ensemble) {
    // First read config
    int timestep_id;
    if (!readEnsembleConfig(num_states, num_ensemble, timestep_id)) {
        return false;
    }

    const char* shm_path = "/dev/shm/ldm_eki_ensemble_data";

    data_fd = open(shm_path, O_RDONLY);
    if (data_fd < 0) {
        fprintf(stderr, "%s[ERROR]%s ", Color::RED, Color::RESET); perror(" Failed to open data");
        return false;
    }

    // Get file size
    struct stat st;
    if (fstat(data_fd, &st) != 0) {
        fprintf(stderr, "%s[ERROR]%s ", Color::RED, Color::RESET); perror(" fstat failed");
        close(data_fd);
        data_fd = -1;
        return false;
    }

    size_t file_size = st.st_size;
    size_t expected_size = sizeof(EnsembleDataHeader) + num_states * num_ensemble * sizeof(float);

    if (file_size != expected_size) {
        std::cerr << Color::RED << "[ERROR] " << Color::RESET << "Size mismatch: file=" << file_size
                  << " bytes, expected=" << expected_size << " bytes" << std::endl;
        close(data_fd);
        data_fd = -1;
        return false;
    }

    // Map entire file
    data_map = mmap(nullptr, file_size, PROT_READ, MAP_SHARED, data_fd, 0);
    if (data_map == MAP_FAILED) {
        fprintf(stderr, "%s[ERROR]%s ", Color::RED, Color::RESET); perror(" mmap failed");
        close(data_fd);
        data_fd = -1;
        return false;
    }

    // Read header
    auto* header = reinterpret_cast<EnsembleDataHeader*>(data_map);

    if (header->status != 1) {
        std::cerr << Color::RED << "[ERROR] " << Color::RESET << "Data not ready (status=" << header->status << ")" << std::endl;
        munmap(data_map, file_size);
        close(data_fd);
        data_map = nullptr;
        data_fd = -1;
        return false;
    }

    if (header->rows != num_states || header->cols != num_ensemble) {
        std::cerr << Color::RED << "[ERROR] " << Color::RESET << "Dimension mismatch: header says " << header->rows
                  << "×" << header->cols << ", config says " << num_states << "×" << num_ensemble << std::endl;
        munmap(data_map, file_size);
        close(data_fd);
        data_map = nullptr;
        data_fd = -1;
        return false;
    }

    // Read data
    float* data_ptr = reinterpret_cast<float*>(
        reinterpret_cast<uint8_t*>(data_map) + sizeof(EnsembleDataHeader)
    );

    size_t data_count = num_states * num_ensemble;
    output.resize(data_count);
    std::memcpy(output.data(), data_ptr, data_count * sizeof(float));

    // Calculate statistics
    float min_val = *std::min_element(output.begin(), output.end());
    float max_val = *std::max_element(output.begin(), output.end());
    float sum = std::accumulate(output.begin(), output.end(), 0.0f);
    float mean_val = sum / data_count;

    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Ensemble states loaded: " << Color::BOLD << num_states << "×" << num_ensemble << Color::RESET
              << " (" << data_count * sizeof(float) / 1024.0 << " KB)\n";
    std::cout << "  Range : [" << min_val << ", " << max_val << "], mean=" << mean_val << "\n";

    // Memory Doctor: Log received ensemble states with iteration from timestep_id
    if (g_memory_doctor.isEnabled()) {
        std::string info = "EKI iteration " + std::to_string(timestep_id) + " from Python";
        g_memory_doctor.logReceivedData("ensemble_states", output.data(),
                                      num_states, num_ensemble, timestep_id, info);
    }

    // Cleanup mapping
    munmap(data_map, file_size);
    close(data_fd);
    data_map = nullptr;
    data_fd = -1;

    return true;
}

// ============================================================================
// Cleanup
// ============================================================================

void EKIReader::cleanup() {
    if (data_map) {
        munmap(data_map, data_size);
        data_map = nullptr;
    }
    if (config_map) {
        munmap(config_map, sizeof(EnsembleConfig));
        config_map = nullptr;
    }
    if (data_fd >= 0) {
        close(data_fd);
        data_fd = -1;
    }
    if (config_fd >= 0) {
        close(config_fd);
        config_fd = -1;
    }
    initialized = false;
}

void EKIReader::unlinkEnsembleSharedMemory() {
    shm_unlink(SHM_ENSEMBLE_CONFIG_NAME);
    shm_unlink(SHM_ENSEMBLE_DATA_NAME);
    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Ensemble shared memory unlinked\n";
}

} // namespace LDM_EKI_IPC
