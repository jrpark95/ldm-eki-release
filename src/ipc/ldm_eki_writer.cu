////////////////////////////////////////////////////////////////////////////////
/// @file    ldm_eki_writer.cu
/// @brief   Implementation of IPC writer for LDM→Python communication
/// @details Implements POSIX shared memory operations for transmitting
///          observation data from C++/CUDA forward model to Python EKI
///          inversion process.
///
/// @author  Juryong Park
/// @date    2025
////////////////////////////////////////////////////////////////////////////////

#include "ldm_eki_writer.cuh"
#include "../core/ldm.cuh"  // For EKIConfig definition
#include "../debug/memory_doctor.cuh"
#include <errno.h>

namespace LDM_EKI_IPC {

////////////////////////////////////////////////////////////////////////////////
// Constructor / Destructor
////////////////////////////////////////////////////////////////////////////////

EKIWriter::EKIWriter()
    : config_fd(-1), data_fd(-1), config_map(nullptr),
      data_map(nullptr), data_size(0), initialized(false) {
}

EKIWriter::~EKIWriter() {
    cleanup();
}

////////////////////////////////////////////////////////////////////////////////
// Initialization
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
/// @brief Initialize shared memory segments with full configuration
///
/// @details
/// Creates two shared memory segments:
/// 1. Configuration segment (128 bytes) - Contains all EKI parameters
/// 2. Data segment (variable size) - Contains observation data
///
/// The configuration segment includes:
/// - Basic dimensions (ensemble size, receptors, timesteps)
/// - Algorithm parameters (iteration, lambda, noise, etc.)
/// - Option flags (adaptive, localized, regularization, etc.)
/// - GPU settings
/// - Memory Doctor mode
///
/// @param[in] eki_config     Complete EKI configuration structure
/// @param[in] num_timesteps  Number of simulation timesteps
///
/// @return true if initialization successful, false on error
///
/// @post /dev/shm/ldm_eki_config created and written
/// @post /dev/shm/ldm_eki_data created and header initialized
////////////////////////////////////////////////////////////////////////////////
bool EKIWriter::initialize(const ::EKIConfig& eki_config, int num_timesteps) {
    if (initialized) {
        std::cerr << "EKIWriter already initialized" << std::endl;
        return false;
    }

    // Calculate data size: header + receptor data
    data_size = sizeof(EKIDataHeader) + eki_config.num_receptors * num_timesteps * sizeof(float);

    // Create config shared memory (now using EKIConfigFull size)
    config_fd = shm_open(SHM_CONFIG_NAME, O_CREAT | O_RDWR, 0660);
    if (config_fd < 0) {
        perror("shm_open config");
        return false;
    }

    if (ftruncate(config_fd, sizeof(EKIConfigFull)) != 0) {
        perror("ftruncate config");
        close(config_fd);
        return false;
    }

    config_map = mmap(nullptr, sizeof(EKIConfigFull), PROT_READ | PROT_WRITE,
                     MAP_SHARED, config_fd, 0);
    if (config_map == MAP_FAILED) {
        perror("mmap config");
        close(config_fd);
        return false;
    }

    // Write full configuration
    auto* config = reinterpret_cast<EKIConfigFull*>(config_map);
    memset(config, 0, sizeof(EKIConfigFull));

    // Basic info
    config->ensemble_size = eki_config.ensemble_size;
    config->num_receptors = eki_config.num_receptors;
    config->num_timesteps = num_timesteps;

    // Algorithm parameters
    config->iteration = eki_config.iteration;
    config->renkf_lambda = eki_config.renkf_lambda;
    config->noise_level = eki_config.noise_level;
    config->time_interval = eki_config.time_interval;
    config->prior_constant = eki_config.prior_constant;

    // Option strings (safe copy with null termination)
    memset(config->perturb_option, 0, 8);
    strncpy(config->perturb_option, eki_config.perturb_option.c_str(), 7);

    memset(config->adaptive_eki, 0, 8);
    strncpy(config->adaptive_eki, eki_config.adaptive_eki.c_str(), 7);

    memset(config->localized_eki, 0, 8);
    strncpy(config->localized_eki, eki_config.localized_eki.c_str(), 7);

    memset(config->regularization, 0, 8);
    strncpy(config->regularization, eki_config.regularization.c_str(), 7);

    memset(config->time_unit, 0, 8);
    strncpy(config->time_unit, eki_config.time_unit.c_str(), 7);

    memset(config->memory_doctor, 0, 8);
    strncpy(config->memory_doctor, eki_config.memory_doctor_mode ? "On" : "Off", 7);

    // Create data shared memory
    data_fd = shm_open(SHM_DATA_NAME, O_CREAT | O_RDWR, 0660);
    if (data_fd < 0) {
        perror("shm_open data");
        munmap(config_map, sizeof(EKIConfigFull));
        close(config_fd);
        return false;
    }

    if (ftruncate(data_fd, data_size) != 0) {
        perror("ftruncate data");
        munmap(config_map, sizeof(EKIConfigFull));
        close(config_fd);
        close(data_fd);
        return false;
    }

    data_map = mmap(nullptr, data_size, PROT_READ | PROT_WRITE,
                   MAP_SHARED, data_fd, 0);
    if (data_map == MAP_FAILED) {
        perror("mmap data");
        munmap(config_map, sizeof(EKIConfigFull));
        close(config_fd);
        close(data_fd);
        return false;
    }

    // Initialize data header
    auto* header = reinterpret_cast<EKIDataHeader*>(data_map);
    header->status = 0;  // Writing status
    header->rows = eki_config.num_receptors;
    header->cols = num_timesteps;

    initialized = true;
    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Writer initialized with full configuration\n";
    std::cout << "  Ensembles      : " << Color::BOLD << eki_config.ensemble_size << Color::RESET << "\n";
    std::cout << "  Receptors      : " << Color::BOLD << eki_config.num_receptors << Color::RESET << "\n";
    std::cout << "  Timesteps      : " << Color::BOLD << num_timesteps << Color::RESET << "\n";
    std::cout << "  Iteration      : " << eki_config.iteration << "\n";
    std::cout << "  Regularization : " << eki_config.regularization << "\n";
    return true;
}

////////////////////////////////////////////////////////////////////////////////
// Write Initial Observations
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
/// @brief Write initial observation matrix to shared memory
///
/// @details
/// Writes the "truth" simulation observations to the data segment.
/// Uses a simple handshake protocol:
/// - status=0: Writing in progress
/// - memcpy data
/// - status=1: Ready for Python to read
///
/// @param[in] observations  Row-major observation matrix [receptors × timesteps]
/// @param[in] rows          Number of receptors
/// @param[in] cols          Number of timesteps
///
/// @return true if write successful, false on error
///
/// @pre initialize() must have been called
/// @post Data written to /dev/shm/ldm_eki_data with status=1
////////////////////////////////////////////////////////////////////////////////
bool EKIWriter::writeObservations(const float* observations, int rows, int cols) {
    if (!initialized) {
        std::cerr << "EKIWriter not initialized" << std::endl;
        return false;
    }

    auto* header = reinterpret_cast<EKIDataHeader*>(data_map);

    // Verify dimensions
    if (rows != header->rows || cols != header->cols) {
        std::cerr << "Dimension mismatch: expected " << header->rows
                  << "x" << header->cols << ", got " << rows << "x" << cols << std::endl;
        return false;
    }

    // Set writing status
    header->status = 0;

    // Copy data
    float* data_ptr = reinterpret_cast<float*>(
        reinterpret_cast<uint8_t*>(data_map) + sizeof(EKIDataHeader)
    );

    memcpy(data_ptr, observations, rows * cols * sizeof(float));

    // Memory Doctor: Log sent data (iteration 0 for initial observations)
    if (g_memory_doctor.isEnabled()) {
        g_memory_doctor.logSentData("initial_observations", observations, rows, cols, 0,
                                   "LDM->Python initial EKI observations");
    }

    // Set ready status
    header->status = 1;

    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Observations written: " << Color::BOLD << rows << "×" << cols << Color::RESET
              << " matrix (" << (rows * cols * sizeof(float)) / 1024.0 << " KB)\n";
    return true;
}

////////////////////////////////////////////////////////////////////////////////
// Ensemble Observations
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
/// @brief Initialize ensemble observation shared memory segments
///
/// @details
/// Creates the configuration segment for ensemble observations.
/// The data segment will be created/truncated on each write.
///
/// @param[in] ensemble_size   Number of ensemble members
/// @param[in] num_receptors   Number of receptors
/// @param[in] num_timesteps   Number of timesteps
///
/// @return true if successful, false on error
///
/// @post /dev/shm/ldm_eki_ensemble_obs_config created and written
////////////////////////////////////////////////////////////////////////////////
bool EKIWriter::initializeEnsembleObservations(int ensemble_size, int num_receptors, int num_timesteps) {
    if (!initialized) {
        std::cerr << "EKIWriter not initialized. Call initialize() first." << std::endl;
        return false;
    }

    // Create ensemble observation config shared memory
    int ens_obs_config_fd = shm_open(SHM_ENSEMBLE_OBS_CONFIG_NAME, O_CREAT | O_RDWR | O_TRUNC, 0666);
    if (ens_obs_config_fd < 0) {
        std::cerr << "Failed to create ensemble obs config shared memory: " << strerror(errno) << std::endl;
        return false;
    }

    // Set size for config (12 bytes)
    if (ftruncate(ens_obs_config_fd, sizeof(EKIConfigBasic)) != 0) {
        std::cerr << "Failed to set ensemble obs config size: " << strerror(errno) << std::endl;
        close(ens_obs_config_fd);
        return false;
    }

    // Map config memory
    void* ens_obs_config_map = mmap(nullptr, sizeof(EKIConfigBasic), PROT_READ | PROT_WRITE, MAP_SHARED, ens_obs_config_fd, 0);
    if (ens_obs_config_map == MAP_FAILED) {
        std::cerr << "Failed to map ensemble obs config memory: " << strerror(errno) << std::endl;
        close(ens_obs_config_fd);
        return false;
    }

    // Write config
    EKIConfigBasic* ens_obs_config = static_cast<EKIConfigBasic*>(ens_obs_config_map);
    ens_obs_config->ensemble_size = ensemble_size;
    ens_obs_config->num_receptors = num_receptors;
    ens_obs_config->num_timesteps = num_timesteps;

    // Unmap config (data will be written separately)
    munmap(ens_obs_config_map, sizeof(EKIConfigBasic));
    close(ens_obs_config_fd);

    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Ensemble observation config: "
              << Color::BOLD << ensemble_size << Color::RESET << " ensembles, "
              << Color::BOLD << num_receptors << Color::RESET << " receptors, "
              << Color::BOLD << num_timesteps << Color::RESET << " timesteps\n";

    return true;
}

////////////////////////////////////////////////////////////////////////////////
/// @brief Write ensemble observation tensor to shared memory
///
/// @details
/// Writes all ensemble member observations for the current iteration.
/// Creates/truncates the data segment each time to ensure fresh data.
///
/// Data Layout:
/// - [ensemble_0][receptor_0][time_0], [ensemble_0][receptor_0][time_1], ...
/// - [ensemble_0][receptor_1][time_0], [ensemble_0][receptor_1][time_1], ...
/// - ...
/// - [ensemble_N][receptor_R][time_T]
///
/// @param[in] observations    Flattened 3D tensor (row-major)
/// @param[in] ensemble_size   Number of ensemble members
/// @param[in] num_receptors   Number of receptors
/// @param[in] num_timesteps   Number of timesteps
/// @param[in] iteration       Current EKI iteration (for logging)
///
/// @return true if write successful, false on error
///
/// @pre initializeEnsembleObservations() must have been called
/// @post /dev/shm/ldm_eki_ensemble_obs_data created/truncated and written
////////////////////////////////////////////////////////////////////////////////
bool EKIWriter::writeEnsembleObservations(const float* observations, int ensemble_size,
                                         int num_receptors, int num_timesteps, int iteration) {
    if (!initialized) {
        std::cerr << "EKIWriter not initialized" << std::endl;
        return false;
    }

    // Calculate data size
    size_t ens_obs_data_size = ensemble_size * num_receptors * num_timesteps * sizeof(float);

    // Create/open data shared memory
    int ens_obs_data_fd = shm_open(SHM_ENSEMBLE_OBS_DATA_NAME, O_CREAT | O_RDWR | O_TRUNC, 0666);
    if (ens_obs_data_fd < 0) {
        std::cerr << "Failed to create ensemble obs data shared memory: " << strerror(errno) << std::endl;
        return false;
    }

    // Set data size
    if (ftruncate(ens_obs_data_fd, ens_obs_data_size) != 0) {
        std::cerr << "Failed to set ensemble obs data size: " << strerror(errno) << std::endl;
        close(ens_obs_data_fd);
        return false;
    }

    // Map data memory
    void* ens_obs_data_map = mmap(nullptr, ens_obs_data_size, PROT_READ | PROT_WRITE, MAP_SHARED, ens_obs_data_fd, 0);
    if (ens_obs_data_map == MAP_FAILED) {
        std::cerr << "Failed to map ensemble obs data memory: " << strerror(errno) << std::endl;
        close(ens_obs_data_fd);
        return false;
    }

    // Write data
    memcpy(ens_obs_data_map, observations, ens_obs_data_size);

    // Calculate statistics for validation
    float min_val = observations[0];
    float max_val = observations[0];
    float sum_val = 0.0f;
    int total_elements = ensemble_size * num_receptors * num_timesteps;

    for (int i = 0; i < total_elements; i++) {
        float val = observations[i];
        if (val < min_val) min_val = val;
        if (val > max_val) max_val = val;
        sum_val += val;
    }

    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Ensemble observations written (" << Color::BOLD << ens_obs_data_size / 1024.0 << " KB" << Color::RESET << ")\n";
    std::cout << "  Shape : [" << ensemble_size << " × " << num_receptors << " × " << num_timesteps << "]\n";
    std::cout << "  Range : [" << min_val << ", " << max_val << "], mean=" << (sum_val / total_elements) << "\n";

    // Memory Doctor: Log sent ensemble observations with iteration
    if (g_memory_doctor.isEnabled()) {
        std::string info = "EKI iteration " + std::to_string(iteration) + ": " + std::to_string(ensemble_size) + " ensembles";
        g_memory_doctor.logSentData("ensemble_observations", observations,
                                   ensemble_size * num_receptors, num_timesteps, iteration, info);
    }

    // Unmap data
    munmap(ens_obs_data_map, ens_obs_data_size);
    close(ens_obs_data_fd);

    return true;
}

////////////////////////////////////////////////////////////////////////////////
// Configuration Retrieval
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
/// @brief Get current basic configuration dimensions
///
/// @param[out] ensemble_size   Number of ensemble members
/// @param[out] num_receptors   Number of receptors
/// @param[out] num_timesteps   Number of timesteps
///
/// @return true if config available, false if not initialized
////////////////////////////////////////////////////////////////////////////////
bool EKIWriter::getConfig(int& ensemble_size, int& num_receptors, int& num_timesteps) {
    if (!initialized) {
        return false;
    }

    auto* config = reinterpret_cast<EKIConfigFull*>(config_map);
    ensemble_size = config->ensemble_size;
    num_receptors = config->num_receptors;
    num_timesteps = config->num_timesteps;
    return true;
}

////////////////////////////////////////////////////////////////////////////////
// Cleanup
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
/// @brief Cleanup resources and unmap memory
///
/// @details
/// Unmaps memory and closes file descriptors, but does NOT unlink the
/// shared memory files from /dev/shm. Use unlinkSharedMemory() for that.
///
/// @post All memory unmapped and file descriptors closed
/// @post initialized flag set to false
////////////////////////////////////////////////////////////////////////////////
void EKIWriter::cleanup() {
    if (data_map) {
        munmap(data_map, data_size);
        data_map = nullptr;
    }
    if (config_map) {
        munmap(config_map, sizeof(EKIConfigFull));
        config_map = nullptr;
    }
    if (data_fd >= 0) {
        close(data_fd);
        data_fd = -1;
    }
    if (config_fd >= 0) {
        close(config_fd);
        config_fd = -1;
    }
    initialized = false;
}

////////////////////////////////////////////////////////////////////////////////
/// @brief Unlink shared memory segments from filesystem
///
/// @details
/// Removes shared memory files from /dev/shm. Should be called at program
/// exit after all readers (Python) have finished.
///
/// Note: Does NOT unlink ensemble observation files because Python needs
/// them after C++ exits.
///
/// @post /dev/shm/ldm_eki_config removed
/// @post /dev/shm/ldm_eki_data removed
////////////////////////////////////////////////////////////////////////////////
void EKIWriter::unlinkSharedMemory() {
    shm_unlink(SHM_CONFIG_NAME);
    shm_unlink(SHM_DATA_NAME);
    // Don't unlink ensemble observation files - Python needs them!
    // shm_unlink(SHM_ENSEMBLE_OBS_CONFIG_NAME);
    // shm_unlink(SHM_ENSEMBLE_OBS_DATA_NAME);
    std::cout << Color::CYAN << "[IPC] " << Color::RESET
              << "Shared memory unlinked (ensemble obs kept for Python)\n";
}

} // namespace LDM_EKI_IPC
